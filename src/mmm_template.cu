#include "hip/hip_runtime.h"
#include<stdio.h>
#include<sys/time.h>
#include<stdlib.h>
#include<iostream>

using namespace std;

//----------------------------------- Structures and Globals---------------------------------------------

typedef struct {
	int dimension1;
	int dimension2;	
} ArrayMetadata2D;

// metadata variables describing dimensionalities of all data structures involved in the computation
ArrayMetadata2D A_MD, B_MD, C_MD;
// pointers for input and output arrays in the host memory  
float *A, *B, *C, *C_CPU;
// pointers for input and output arrays in the device memory (NVIDIA DRAM)
float *A_GPU, *B_GPU, *C_GPU;

//----------------------------------- host function definitions -----------------------------------------

void allocateAndInitializeAB();
void computeCpuMMM();/
void copyMatricesToGPU();
void copyResultFromGPU();
void compareHostAndGpuOutput();
void die(const char *error); 
void check_error(hipError_t e);
__global__ void kernel(float * A_GPU, float * B_GPU, float * C_GPU, ArrayMetaData2D A_gpu_md, ArrayMetaData2D B_gpu_md);
//----------------------------------- CUDA function definitions -----------------------------------------

#define BLOCK_SIZE 10

//-------------------------------------------------------------------------------------------------------
int main(int argc, char **argv) {
	
	A_MD.dimension1 = (argc > 1) ? atoi(argv[1]) : 100;
	A_MD.dimension2 = (argc > 2) ? atoi(argv[2]) : A_MD.dimension1;
	B_MD.dimension1 = (argc > 3) ? atoi(argv[3]) : A_MD.dimension2;
	B_MD.dimension2 = (argc > 4) ? atoi(argv[4]) : B_MD.dimension1;
	C_MD.dimension1 = A_MD.dimension1;
	C_MD.dimension2 = B_MD.dimension2;

	printf("Matrix A is %d-by-%d\n", A_MD.dimension1, A_MD.dimension2);
	printf("Matrix B is %d-by-%d\n", B_MD.dimension1, B_MD.dimension2);
	printf("Matrix C is %d-by-%d\n", C_MD.dimension1, C_MD.dimension2);

	allocateAndInitializeAB();

	// matrix matrix multiplication in the CPU
	clock_t start = clock();	
	computeCpuMMM();
	clock_t end = clock();
        double elapsed = (end - start) / (double) CLOCKS_PER_SEC;
        printf("Computation time in the CPU: %f seconds\n", elapsed);
	

	cuda_mat_mul();	
	copyMatricesToGPU();
	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
	// width of b / BS, height of A / BS
	dim3 dimGrid(B_MD.dimension1/ BLOCK_SIZE, A_MD.dimesion2 / BLOCK_SIZE);
	kernel <<< dimGrid, dimBlock >>>  (A_GPU, B_GPU, C_GPU, A_MD, B_MD);

	return 0;
}

// allocate and initialize A and B using a random number generator
void allocateAndInitializeAB() {
	
	size_t sizeofA = A_MD.dimension1 * A_MD.dimension2 * sizeof(float);
	A = (float*) malloc(sizeofA);
	
	srand(time(NULL));
  	for (int i = 0; i < A_MD.dimension1; i++) {
		for (int j = 0; j < A_MD.dimension2; j++) {
			int index = i * A_MD.dimension2 + j;
			A[index] = (rand() % 1000) * 0.001; 
		}
	}
	
	size_t sizeofB = B_MD.dimension1 * B_MD.dimension2 * sizeof(float);
	B = (float*) malloc(sizeofB);
  	for (int i = 0; i < B_MD.dimension1; i++) {
		for (int j = 0; j < B_MD.dimension2; j++) {
			int index = i * B_MD.dimension2 + j;
			B[index] = (rand() % 1000) * 0.001; 
		}
	}
}

// allocate memory in the GPU for all matrices, and copy A and B content from the host CPU memory to the GPU memory
void copyMatricesToGPU() {
	
	size_t sizeofA = A_MD.dimension1 * A_MD.dimension2 * sizeof(float);
	check_error(hipMalloc((void **) &A_GPU, sizeofA));
	check_error(hipMemcpy(A_GPU, A, sizeofA, hipMemcpyHostToDevice));
	
	size_t sizeofB = B_MD.dimension1 * B_MD.dimension2 * sizeof(float);
	check_error(hipMalloc((void **) &B_GPU, sizeofB));
	check_error(hipMemcpy(B_GPU, B, sizeofB, hipMemcpyHostToDevice));
	
	size_t sizeofC = C_MD.dimension1 * C_MD.dimension2 * sizeof(float);
	check_error(hipMalloc((void **) &C_GPU, sizeofC));
}

// copy results from C_GPU which is in GPU card memory to C_CPU which is in the host CPU for result comparison
void copyResultFromGPU() {
	size_t sizeofC = C_MD.dimension1 * C_MD.dimension2 * sizeof(float);
	C_CPU = (float*) malloc(sizeofC);
	check_error(hipMemcpy(C_CPU, C_GPU, sizeofC, hipMemcpyDeviceToHost));
}

// do a straightforward matrix-matrix multiplication in the CPU
// notice that this implementation can be massively improved in the CPU by doing proper cache blocking but we are
// not providing you the efficient CPU implementation as that reveals too much about the ideal GPU implementation
void computeCpuMMM() {
	
	// allocate the result matrix for the CPU computation
	size_t sizeofC = C_MD.dimension1 * C_MD.dimension2 * sizeof(float);
	C = (float*) malloc(sizeofC);
	
	// compute C[i][j] as the sum of A[i][k] * B[k][j] for all columns k of A
	for (int i = 0; i < A_MD.dimension1; i++) {
		int a_i = i * A_MD.dimension2;
		int c_i = i * C_MD.dimension2;
		for (int j = 0; j < B_MD.dimension2; j++) {
			int c_index = c_i + j;
			C[c_index] = 0;
			for (int k = 0; k < B_MD.dimension1; k++) {
				int a_index = a_i + k;
				int b_index = k * B_MD.dimension2 + j;
				C[c_index] += A[a_index] * B[b_index];
			}
		}
	}
}

__global__ void kernel(float * A_GPU, float * B_GPU, float * C_GPU, ArrayMetaData2D A_gpu_md, ArrayMetaData2D B_gpu_md) {
	////////////////////////////////////
	// Marcus's idea of how it should work
	const int blockY = blockIdx.y; // the global block indexes
	const int blockX = blockIdx.x;	
	
	// Get the reference to C starting at the row and column
	// Essentially this is the whole block
	// I've probably f'ed up the index
	float * C_block = &C_GPU[A_gpu_md.dimension2 * blockY * BLOCK_SIZE + blockX * BLOCK_SIZE]
	
	const int sub_row = threadIdx.y; // valued from 0:blocksize-1
	const int sub_col = threadIdx.x; // valued from 0:blocksize-1
	
	// Th value we're going to shove into the final array
	volatile float my_final_value = 0.0f;
	
	// Let's loop over each block!
	for (int i = 0; i < A_gpu_md.dimension2 / BLOCK_SIZE; m++) {
		// Get the sub block
		float * A_block = A_GPU[];
		float * B_block = B_GPU[];
	
		// Here's all the shared memory we'll need
		__shared__ float sharedA[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float sharedB[BLOCK_SIZE][BLOCK_SIZE];

		// Fill in that shared array with my column
		sharedA[sub_row][sub_col] = A_block[];
		sharedB[sub_row][sub_col] = B_block[];

		// Sum up all the elements that go from 0:BLOCKSIZE
		// So the row of A and the column of B for 0 to BLOCKSIZE
		for (int j = 0; j < BLOCK_SIZE; e++) my_final_value += sharedA[sub_row][j] * sharedB[j][sub_col]
	}
	
	C_block[sub_row * B_gpu_md.dimension1 + sub_col] = my_final_value;
	//
	////////////////////////////////////////////////	
	int srow = 0;
	int scol = 0;
	int threadId = blockDim.x * blockIdx.x + threadIdx.x;
	const int sizeOfWork = 10;
	const int sizeOfBlock = 100;
	// Where to start in the GPU matrix
	int mIndex = threadId * sizeOfWork;

	// copy the submatrix into shared memory
	__shared__ float blockA[10][10]; 
	__shared__ float blockB[10][10]; 
	int blockIndex = mIndex;
	for (int i = 0; i < sizeOfBlock; i++) {
		blockA[srow][scol] = A_GPU[blockIndex];
		blockB[srow][scol] = B_GPU[blockIndex];
		// Jump a row when finished copying column
		if (i == sizeOfWork) {
			srow++;
			blockIndex *= sizeOfWork;
		}
		scol++;
	}

	// Compute a partial row of C
	int aRow = threadId;
	int cIndex = mIndex;
	// TODO: Transpose B for better load times
	// Will need to switch order to keep coalesced 

	// Multiply a row of A 
	for (int aCol = 0; aCol < sizeOfWork; aCol++) {
		// with each column of B
		for (int bCol = 0; bCol < sizeOfWork; bCol++) {
			float cell = 1;
			for (int bRow = 0; bRow < sizeOfWork; bRow++) {
				cell += (blockA[aRow][aCol] * blockB[bRow][bCol]);
			}
			// Store the result in C
			C_GPU[cIndex] = cell;
			cIndex++;
		}
	}



}



// function to determine if the GPU computation is done correctly by comparing the output from the GPU with that
// from the CPU
void compareHostAndGpuOutput() {
	int totalElements = C_MD.dimension1 * C_MD.dimension2;
	int missmatchCount = 0;
	for (int i = 0; i < totalElements; i++) {
		if (fabs(C[i] - C_CPU[i]) > 0.01) {
			missmatchCount++;
			printf("mismatch at index %i: %f\t%f\n", i, C[i], C_CPU[i]);
		}
	}
	if (missmatchCount > 0) {
		printf("Computation is incorrect: outputs do not match in %d indexes\n", missmatchCount);
	} else {
		printf("Computation is correct: CPU and GPU outputs match\n");
	}
}

// Prints the specified error message and then exits
void die(const char *error) {
        printf("%s", error);
        exit(1);
}

// If the specified error code refers to a real error, report it and quit the program
void check_error(hipError_t e) {
        if (e != hipSuccess) {
                printf("\nCUDA error: %s\n", hipGetErrorString(e));
                exit(1);
        }
}

